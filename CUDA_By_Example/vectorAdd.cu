
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 10

__global__ void vectorAdd(int *a, int *b, int *c){
	int tid = blockIdx.x;
	c[tid] = a[tid]+b[tid];
}

int main(){
	int a[N], b[N], c[N];
	int *dev_a, *dev_b, *dev_c;
	
	for(int i = 0; i < N; i++){
		*(a+i) = i;
		*(a+i) = i*i;
	}

	hipMalloc( (void**)&dev_a, N*sizeof(int) );
	hipMalloc( (void**)&dev_b, N*sizeof(int) );
	hipMalloc( (void**)&dev_c, N*sizeof(int) );

	hipMemcpy( dev_a, a, N*sizeof(int), hipMemcpyHostToDevice );
	hipMemcpy( dev_b, b, N*sizeof(int), hipMemcpyHostToDevice );

	vectorAdd<<<N, 1>>>(a, b, c);

	hipMemcpy( c, dev_c, N*sizeof(int), hipMemcpyDeviceToHost );
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	for(int i = 0; i < N; i++){
		printf("%d + %d = %d\n", a[i], b[i], c[i]);
	}

	return 0;
}
