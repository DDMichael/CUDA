
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add( int a, int b, int *c ){
	*c = a + b;
}

int main(void){
	int c;
	int *dev_c; // pointer to device
	hipMalloc( (void**)&dev_c, sizeof(int) );
	add<<<1, 1>>>( 2, 8, dev_c);
	hipMemcpy( &c, dev_c, sizeof(int), hipMemcpyDeviceToHost);
	printf( "2 + 7 = %d\n", c );
	hipFree( dev_c );
	return 0;
}
